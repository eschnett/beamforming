#include "hip/hip_runtime.h"
// -*-c++-*-
// Beamforming with CUDA

#include "arraysizes.hxx"
#include "icomplex4.hxx"

#include <mma.h>

#include <iostream>

using namespace std;

using namespace nvcuda;
using namespace nvcuda::wmma;

////////////////////////////////////////////////////////////////////////////////

// Index spaces

// Idea:
//
// Create "index spaces" that define the set of indices over which a kernel is applied.
//
// Index spaces cannot be split, but they can be combined.
//
// They can be combined to index arrays (and to define arrays).
//
// They can be combined into threads, warps, blocks, loop iterations, or be "explicit" (as in complex number components).
//
// There are checks that the mappings (to arrays and threads etc.) are complete, and don't overlap.
//
// Bank conflicts for shared and global memory should fall out naturally.

// Let's define the index spaces we are using:

// Note: We ignore frequencies here since we can map them to blocks.

// CUDA:
//
// 32 threads per warp
// 32 warps
//
// matrix m = 8    (beams)
// matrix n = 8    (times)
// matrix k = 32   (dishes)

// Input and output arrays:
//
// ntimes = 32768      (per chunk)
// nfrequencies = 32   (per GPU)
// ndishes = 512
// npolarizations = 2
// nbeams = 96
// ncomplex = 2        (complex number components)

// load_A:
//
// 3 iterations for beams
// 2 iterations for dishes
// 4 warps for beams
// 8 warps for dishes
// 32 threads for matrices
// 32 k matrix elements for dishes
// 8 m matrix elements for beams
// 2 explicit for complex numbers
//
// beam = (beam iteration) * (beam warp) * (beam m matrix element)
// dish = (dish iteration) * (dish warp) * (dish k matrix element)
// complex = (explicit)
//
// input: A[frequency][beam][dish][complex]
// output: A_register[complex][beam / A_register_beam_divisor][dish / A_register_dish_divisor]
//         [beam % A_register_beam_matrix_modulo][dish % A_register_dish_matrix_modulo]
// A_register_beam_divisor = (# beam warp) * (# beam m matrix element)
// A_register_dish_divisor = (# dish warp) * (# dish k matrix element)
// A_register_beam_matrix_modulo = (# beam m matrix element)
// A_register_dish_matrix_modulo = (# dish k matrix element)
//
// TODO: Use dish' instead of dish

// shuffle_E:
//
// 512 iterations (outer) for times
// 4 iterations (inner) for times
// 16 warps for times
// 2 warps for dishes
// 32 threads for dishes
// 4 explicit (outer) for dishes
// 2 explicit (inner) for dishes
// 2 explicit for polarization
// 2 explicit for complex numbers
//
// time = (time iteration outer) * (time iteration inner) * (time warp)
// dish = (dish warp) * (dish explicit outer) * (dish thread) * (dish explicit inner)
// polarization = (explicit)
// complex = (explicit)
// dish' = (dish warp) * (dish thread) * (dish explicit inner) * (dish explicit outer)
//
// input: E[time][frequency][dish][polarization][complex]
// output: E_shared[complex][time % E_shared_time_modulo][polarization][dish' + padding]
// E_shared_time_modulo = (# time warp)
//
// E layout:
//   dish warp
//     dish explicit outer
//        dish thread
//              dish explicit inner
//               polarization explicit
//                complex explicit
//   d dd ddddd dpc
//   8 76 54321 000
//
// Extract complex components:
//   cd d ddddd dpd
//   08 7 54321 006
//
// Extract polarization:
//   pcd ddddd ddd
//   008 54321 076

// compute_Ju:
//
// 512 iterations (outer) for times
// 4 iterations (inner) for times
// 4 iterations (inner2) for times
// 3 iterations for beams
// 2 iterations for dishes'
// 4 warps for beams
// 8 warps for dishes'
// 8 m matrix elements for beams
// 4 n matrix elements for time
// 2 n matrix elements for polarization
// 32 k matrix elements for dishes
// 2 explicit for complex numbers
//
// time = (time iteration outer) * (time iteration inner) * (time iteration inner2) * (time n matrix element)
// beam = (beam iteration) * (beam warp) * (beam m matrix element)
// dish' = (dish' warp) * (dish' iteration) * (dish' matrix element)
// polarization = (polarization n matrix element)
// complex = explicit
//
// input: A_register[complex][beam][dish]   [beam][dish]
// input: E_shared[complex][time][polarization][dish' + padding][complex]
// output: Ju_shared[dish' / Ju_shared_dish'_divisor][beam][time / Ju_shared_time_divisor %  Ju_shared_time_modulo]
//                  [polarization][complex]
// u_shared_dish'_divisor = (# dish' warp)
// Ju_shared_time_divisor = (# time n matrix elements)
// Ju_shared_time_modulo = (# time iteration inner2)

// reduce_to_J:
//
// 512 iterations (outer) for times
// 4 iterations (inner) for times
// 4 iterations (inner2) for times
// 12 warps for beams (other warps unused)
// 8 threads for beams
// 4 threads for times
// 8 iterations over dishes' (reduction)
// 2 explicit for polarization
// 2 explicit for complex numbers
//
// beam = (beam warp) * (beam thread)
// time = (time iteration (outer)) * (time iteration (inner)) * (time iteration (inner2)) * (time thread)
//
// input: Ju_shared[dish' / Ju_shared_dish'_divisor][beam][time % Ju_shared_time_modulo][polarization][complex]
// output: J_shared[beam][time % J_shared_time_modulo + padding][polarization][complex]
// J_shared_time_modulo = (# iterations (inner) for times) * (# iterations (inner2) for times) * (# threads for times)

// transpose_J:
//
// 512 iterations (outer) for times
// 3 iterations for beams
// 32 warps for beams
// 16 threads for time
// 2 threads for polarization
// 2 explicit for complex
//
// time = (time iteration (outer)) * (time thread)
// beam = (beam iteration) * (beam warp)
//
// input: J_shared[beam][time % J_shared_time_modulo + padding][polarization][complex]
// output: J[beam][frequency][polarization][time][complex]

////////////////////////////////////////////////////////////////////////////////

// Helper functions

__device__ int32_t clamp(int32_t i, int32_t i0, int32_t i1) { return max(i0, min(i1, i)); }

__device__ int32_t extract_real(const int32_t x0, const int32_t x1) {
  return ((uint32_t)(x0 & 0xf0f0f0f0U) >> 4) | (x1 & 0xf0f0f0f0U);
}

__device__ int32_t extract_imag(const int32_t x0, const int32_t x1) {
  return (x0 & 0x0f0f0f0fU) | ((uint32_t)(x1 & 0x0f0f0f0fU) << 4);
}

////////////////////////////////////////////////////////////////////////////////

// CUDA related constants

constexpr size_t num_reals_int = 8; // number of real numbers (complex number components) per 32-bit int

constexpr size_t num_threads = 32; // 32 threads per warp
constexpr size_t num_warps = 32;   // 32 warps

constexpr size_t num_m_elements = 8;  // (mostly) beams
constexpr size_t num_n_elements = 8;  // (mostly) times
constexpr size_t num_k_elements = 32; // (mostly) dishes

// Import algorithm settings

constexpr size_t num_times = ntimes;
constexpr size_t num_frequencies = nfrequencies;
constexpr size_t num_dishes = ndishes;
constexpr size_t num_polarizations = npolarizations;
constexpr size_t num_beams = nbeams;
constexpr size_t num_complex = ncomplex;

constexpr size_t num_dishes_prime = num_dishes;

////////////////////////////////////////////////////////////////////////////////

namespace load_A {

constexpr size_t num_beam_iterations = 3;
constexpr size_t num_dish_iterations = 2;
constexpr size_t num_beam_warps = 4;
constexpr size_t num_dish_warps = 8;
constexpr size_t num_dish_k_elements = 32;
constexpr size_t num_beam_m_elements = 8;
constexpr size_t num_complex_explicit = 2;

static_assert(num_beam_warps * num_dish_warps == num_warps);
static_assert(num_beam_iterations * num_beam_warps * num_beam_m_elements == num_beams);
static_assert(num_dish_iterations * num_dish_warps * num_dish_k_elements == num_dishes);

constexpr size_t A_register_beam_divisor = num_beam_warps * num_beam_m_elements;
constexpr size_t A_register_dish_divisor = num_dish_warps * num_dish_k_elements;
constexpr size_t A_register_beam_matrix_modulo = num_beam_m_elements;
constexpr size_t A_register_dish_matrix_modulo = num_dish_k_elements;

using A_register_t = fragment<wmma::matrix_a, num_m_elements, num_n_elements, num_k_elements, experimental::precision::s4,
                              row_major>[ncomplex][nbeams / A_register_beam_divisor][ndishes / A_register_dish_divisor];

__device__ void load_A(A_register_t &restrict A_register, const ucomplex4 *restrict const A_array, const size_t frequency) {
  const size_t beam_warp = threadIdx.y / num_dish_warps;
  const size_t dish_warp = threadIdx.y % num_dish_warps;
  for (size_t beam_iteration = 0; beam_iteration < num_beam_iterations; ++beam_iteration) {
    for (size_t dish_iteration = 0; dish_iteration < num_dish_iterations; ++dish_iteration) {
      const size_t beam = (beam_iteration * num_beam_warps + beam_warp) * num_beam_m_elements;
      const size_t dish = (dish_iteration * num_dish_warps + dish_warp) * num_dish_k_elements;
      assert(beam < nbeams);
      assert(dish < ndishes);

      // Note: This is the wrong ordering for A; need to shuffle dish
      // indices the same way as for E

      fragment<wmma::matrix_a, num_m_elements, num_n_elements, num_k_elements, experimental::precision::s4, row_major> A0[ncomplex];
      for (size_t c = 0; c < ncomplex; ++c) {
        // TOOD: Use __ldcs
        // Load 2 consecutive sets of elements of A
        load_matrix_sync(A0[c], &A_array[Alinear(frequency, beam, dish + c * num_dish_k_elements / 2, 0) / 2],
                         Alinear(0, 0, 1, 0) / 2);
      }

      assert(beam / A_register_beam_divisor == beam_iteration);
      assert(dish / A_register_dish_divisor == dish_iteration);
      static_assert(A_register[0][beam_iteration][dish_iteration].num_storage_elements == 1);
      for (int i = 0; i < A_register[0][beam_iteration][dish_iteration].num_storage_elements; ++i) {
        // Extract complex components and remove bias
        A_register[0][beam_iteration][dish_iteration].x[i] = extract_real(A0[0].x[i], A0[1].x[i]) ^ 0x88888888U;
        A_register[1][beam_iteration][dish_iteration].x[i] = extract_imag(A0[0].x[i], A0[1].x[i]) ^ 0x88888888U;
      }
    }
  }
}

} // namespace load_A

using load_A::num_beam_m_elements;
using load_A::num_dish_k_elements;

using load_A::A_register_beam_divisor;
using load_A::A_register_beam_matrix_modulo;
using load_A::A_register_dish_divisor;
using load_A::A_register_dish_matrix_modulo;

using load_A::A_register_t;

////////////////////////////////////////////////////////////////////////////////

namespace shuffle_E {

constexpr size_t num_time_iterations_outer = 512;
constexpr size_t num_time_iterations_inner = 4;
constexpr size_t num_time_warps = 16;
constexpr size_t num_dish_warps = 2;
constexpr size_t num_dish_threads = 32;
constexpr size_t num_dish_explicit_outer = 4;
constexpr size_t num_dish_explicit_inner = 2;
constexpr size_t num_polarization_explicit = 2;
constexpr size_t num_complex_explicit = 2;

static_assert(num_time_warps * num_dish_warps == num_warps);
static_assert(num_dish_threads == num_threads);
static_assert(num_time_iterations_outer * num_time_iterations_inner * num_time_warps == num_times);
static_assert(num_dish_warps * num_dish_threads * num_dish_explicit_outer * num_dish_explicit_inner == num_dishes);

constexpr size_t E_shared_time_modulo = num_time_warps;
constexpr size_t E_shared_dish_prime_divisor = num_reals_int;
constexpr size_t E_shared_padding = 4;

using E_shared_t = uint32_t[num_complex][E_shared_time_modulo][num_polarizations]
                           [num_dishes_prime / E_shared_dish_prime_divisor + E_shared_padding];

__device__ void shuffle_E(E_shared_t &restrict E_shared, const ucomplex4 *restrict const E_array, const size_t frequency,
                          const size_t time_iteration_outer, const size_t time_iteration_inner) {
  const size_t time_warp = threadIdx.y / num_dish_warps;
  const size_t dish_warp = threadIdx.y % num_dish_warps;
  const size_t dish_thread = threadIdx.x % num_dish_threads;
  const size_t time = (time_iteration_outer * num_time_iterations_inner + time_iteration_inner) * num_time_warps + time_warp;
  const size_t dish0 = (dish_warp * num_dish_explicit_outer * num_dish_threads + dish_thread) * num_dish_explicit_inner;
  const size_t dish0_prime = (dish_warp * num_dish_threads + dish_thread) * num_dish_explicit_inner * num_dish_explicit_outer;
  assert(time < num_times);
  assert(dish0 < num_dishes);
  assert(dish0_prime < num_dishes);

  // Load E-field from global memory
  // Note: These are not yet split into polarizations complex components. `p` and `c` are the "outer" explicit indices.
  uint32_t E0[num_polarizations][num_complex];
  for (size_t p = 0; p < num_polarizations; ++p) {
    for (size_t c = 0; c < num_complex; ++c) {
      static_assert(num_dish_explicit_outer == num_polarizations * num_complex);
      static_assert(num_dish_explicit_outer * num_dish_explicit_inner == num_reals_int);
      const size_t dish = dish0 + (p * num_complex + c) * num_dish_threads * num_dish_explicit_inner;
      assert(dish < num_dishes);
      assert(uintptr_t(&E_array[Elinear(time, frequency, dish, 0, 0) / 2]) % sizeof(uint32_t) == 0);
      // TOOD: Use __ldcs
      // TOOD: Use __stcs for J array
      E0[p][c] = *(const uint32_t *)&E_array[Elinear(time, frequency, dish, 0, 0) / 2];
    }
  }

  // First we split out the complex components and remove the bias
  uint32_t E1[num_polarizations][num_complex];
  for (size_t p = 0; p < num_polarizations; ++p) {
    E1[p][0] = extract_real(E0[p][0], E0[p][1]) ^ 0x88888888U;
    E1[p][1] = extract_imag(E0[p][0], E0[p][1]) ^ 0x88888888U;
  }

  // Next we separate the polarizations
  uint32_t E2[num_polarizations][num_complex];
  for (size_t c = 0; c < num_complex; ++c) {
    E2[0][c] = __byte_perm(E1[0][c], E1[1][c], 0x6420);
    E2[1][c] = __byte_perm(E1[0][c], E1[1][c], 0x7531);
  }

  // Store into shared memory
  for (size_t c = 0; c < num_complex; ++c) {
    for (size_t p = 0; p < num_polarizations; ++p) {
      const size_t dish_prime = dish0_prime;
      assert(dish_prime < num_dishes_prime);
      E_shared[c][time % shuffle_E::E_shared_time_modulo][p][dish_prime / shuffle_E::E_shared_dish_prime_divisor] = E2[p][c];
    }
  }
}

} // namespace shuffle_E

using shuffle_E::num_time_iterations_inner;
using shuffle_E::num_time_iterations_outer;

using shuffle_E::E_shared_dish_prime_divisor;
using shuffle_E::E_shared_padding;
using shuffle_E::E_shared_time_modulo;

using shuffle_E::E_shared_t;

////////////////////////////////////////////////////////////////////////////////

namespace compute_Ju {

constexpr size_t num_time_iterations_inner2 = 4;
constexpr size_t num_beam_iterations = 3;
constexpr size_t num_dish_prime_iterations = 2;
constexpr size_t num_beam_warps = 4;
constexpr size_t num_dish_prime_warps = 8;
constexpr size_t num_beam_m_elements = 8;
constexpr size_t num_time_n_elements = 4;
constexpr size_t num_polarization_n_elements = 2;
constexpr size_t num_dish_prime_k_elements = 32;
constexpr size_t num_complex_explicit = 2;

static_assert(num_beam_warps * num_dish_prime_warps == num_warps);
static_assert(num_time_iterations_outer * num_time_iterations_inner * num_time_iterations_inner2 * num_time_n_elements ==
              num_times);
static_assert(num_beam_iterations * num_beam_warps * num_beam_m_elements == num_beams);
static_assert(num_dish_prime_iterations * num_dish_prime_warps * num_dish_prime_k_elements == num_dishes);

constexpr size_t Ju_shared_dish_prime_divisor = num_dish_prime_iterations * num_dish_prime_k_elements;
constexpr size_t Ju_shared_time_divisor = num_time_n_elements;
constexpr size_t Ju_shared_time_modulo = num_time_iterations_inner2;
using Ju_shared_t =
    uint32_t[num_dishes_prime / Ju_shared_dish_prime_divisor][num_beams][Ju_shared_time_modulo]; // [polarization][complex]

__device__ void compute_Ju(Ju_shared_t &restrict Ju_shared, const A_register_t &restrict A_register,
                           const E_shared_t &restrict E_shared, const float *restrict const G_array, const size_t frequency,
                           const size_t time_iteration_outer, const size_t time_iteration_inner,
                           const size_t time_iteration_inner2) {
  const size_t beam_warp = threadIdx.y / num_dish_prime_warps;
  const size_t dish_prime_warp = threadIdx.y % num_dish_prime_warps;

  // Load E-field from shared memory
  // wmma::B[k][n]   (must be row major)
  fragment<wmma::matrix_b, num_m_elements, num_n_elements, num_k_elements, experimental::precision::s4, col_major>
      E[num_complex][num_dish_prime_iterations];

  for (size_t c = 0; c < num_complex; ++c) {
    for (size_t dish_prime_iteration = 0; dish_prime_iteration < num_dish_prime_iterations; ++dish_prime_iteration) {
      const size_t time0 = ((time_iteration_outer * num_time_iterations_inner + time_iteration_inner) * num_time_iterations_inner2 +
                            time_iteration_inner2) *
                           num_time_n_elements;
      const size_t dish_prime0 = (dish_prime_warp * num_dish_prime_iterations + dish_prime_iteration) * num_dish_prime_k_elements;
      load_matrix_sync(
          E[c][dish_prime_iteration],
          &E_shared[c][time0 / Ju_shared_time_divisor % Ju_shared_time_modulo][0][dish_prime0 / E_shared_dish_prime_divisor],
          (&E_shared[0][0][1][0] - &E_shared[0][0][0][0]) * num_reals_int);
    }
  }

  for (size_t beam_iteration = 0; beam_iteration < num_beam_iterations; ++beam_iteration) {
    const size_t time0 = ((time_iteration_outer * num_time_iterations_inner + time_iteration_inner) * num_time_iterations_inner2 +
                          time_iteration_inner2) *
                         num_time_n_elements;
    const size_t beam0 = (beam_iteration * num_beam_warps + beam_warp) * num_beam_m_elements;
    const size_t dish_prime0 = dish_prime_warp * num_dish_prime_iterations * num_dish_k_elements;

    fragment<wmma::accumulator, num_m_elements, num_n_elements, num_k_elements, int32_t> JurePos, JureNeg, JuimPos;

    // Initialize Ju
    fill_fragment(JurePos, 0);
    fill_fragment(JureNeg, 0);
    fill_fragment(JuimPos, 0);

    // Multiply
    for (size_t dish_prime_iteration = 0; dish_prime_iteration < num_dish_prime_iterations; ++dish_prime_iteration) {
      mma_sync(JurePos, A_register[0][beam_iteration][dish_prime_iteration], E[0][dish_prime_iteration], JurePos);
      mma_sync(JureNeg, A_register[1][beam_iteration][dish_prime_iteration], E[1][dish_prime_iteration], JureNeg);
      mma_sync(JuimPos, A_register[0][beam_iteration][dish_prime_iteration], E[1][dish_prime_iteration], JuimPos);
      mma_sync(JuimPos, A_register[1][beam_iteration][dish_prime_iteration], E[0][dish_prime_iteration], JuimPos);
    }

    // Extract result from Ju matrix
    int8_t Ju8[num_polarizations][num_complex];
    static_assert(JurePos.num_storage_elements == npolarizations);
    for (size_t i = 0; i < JurePos.num_storage_elements; ++i) {
      const size_t element = threadIdx.x * JurePos.num_storage_elements + i;
      // const size_t time = time0 + element / num_m_elements;
      const size_t beam = beam0 + element % num_m_elements / num_polarizations;
      const size_t p = element % num_m_elements % num_polarizations;
      // Combine positive and negative J values, and reduce from 32 to 16 bits
      int32_t Ju[num_complex];
      Ju[0] = JurePos.x[i] - JureNeg.x[i];
      Ju[1] = JuimPos.x[i];
      for (size_t c = 0; c < num_complex; ++c) {
        assert(uintptr_t(&G_array[Glinear(frequency, beam)]) % sizeof(float) == 0);
        const float G = G_array[Glinear(frequency, beam)];
        Ju8[p][c] = clamp(int32_t(lrintf(G * float(Ju[c]))), -127, 127);
      }
    }
    // CUDA is little endian
    // TODO: Use make_char4
    // TODO: Use char4/uchar4 instead of uint32_t?
    const uint32_t Ju8all =
        ((uint32_t)Ju8[0][0]) | ((uint32_t)Ju8[0][1] << 8) | ((uint32_t)Ju8[1][0] << 16) | ((uint32_t)Ju8[1][1] << 24);

    const size_t element0 = threadIdx.x * JurePos.num_storage_elements;
    const size_t beam = beam0 + element0 % num_m_elements / num_polarizations;
    assert(dish_prime0 / Ju_shared_dish_prime_divisor == dish_prime_warp);
    Ju_shared[dish_prime_warp][beam][time0 / Ju_shared_time_divisor % Ju_shared_time_modulo] = Ju8all;
  }
}
} // namespace compute_Ju

using compute_Ju::num_time_iterations_inner2;

using compute_Ju::Ju_shared_dish_prime_divisor;
using compute_Ju::Ju_shared_time_divisor;
using compute_Ju::Ju_shared_time_modulo;

using compute_Ju::Ju_shared_t;

////////////////////////////////////////////////////////////////////////////////

namespace reduce_to_J {

constexpr size_t num_beam_warps = 12; // other warps are unused
constexpr size_t num_beam_threads = 8;
constexpr size_t num_time_threads = 4;
constexpr size_t num_dish_prime_iterations = 8; // for reduction
constexpr size_t num_polarizations_explicit = 2;
constexpr size_t num_complex_explicit = 2;

static_assert(num_beam_warps <= num_warps);
static_assert(num_beam_threads * num_time_threads == num_threads);
static_assert(num_time_iterations_outer * num_time_iterations_inner * num_time_iterations_inner2 * num_time_threads == num_times);
static_assert(num_beam_warps * num_beam_threads == num_beams);
static_assert(num_dish_prime_iterations == num_dishes_prime / Ju_shared_dish_prime_divisor);

constexpr size_t J_shared_time_modulo = num_time_iterations_inner * num_time_iterations_inner2 * num_time_threads;
constexpr size_t J_shared_padding = 4;

using J_shared_t = uint16_t[num_beams][J_shared_time_modulo + J_shared_padding]; // [polarization][complex]

__device__ void reduce_to_J(J_shared_t &restrict J_shared, const Ju_shared_t &restrict Ju_shared, const size_t time_iteration_outer,
                            const size_t time_iteration_inner, const size_t time_iteration_inner2) {
  const size_t beam_warp = threadIdx.y;
  if (beam_warp < num_beam_warps) {
    // Other warps are unused
    const size_t beam_thread = threadIdx.x / num_time_threads;
    const size_t time_thread = threadIdx.x % num_time_threads;
    const size_t beam = beam_warp * num_beam_threads + beam_thread;
    const size_t time = ((time_iteration_outer * num_time_iterations_inner + time_iteration_inner) * num_time_iterations_inner2 +
                         time_iteration_inner2) *
                            num_time_threads +
                        time_thread;

    // TODO: Vectorize this
    int8_t J[num_polarizations][num_complex];
    for (size_t p = 0; p < num_polarizations; ++p) {
      for (size_t c = 0; c < num_complex; ++c) {
        J[p][c] = 0;
      }
    }
    for (size_t dish_prime_iteration = 0; dish_prime_iteration < num_dish_prime_iterations; ++dish_prime_iteration) {
      for (size_t p = 0; p < num_polarizations; ++p) {
        for (size_t c = 0; c < num_complex; ++c) {
          uint32_t Ju = Ju_shared[dish_prime_iteration][beam][time / Ju_shared_time_divisor % Ju_shared_time_modulo];
          J[p][c] += int8_t((Ju >> (8 * (p * num_complex + c))) & 0xffU);
        }
      }
    }
    // Convert to 4 bits and add bias
    uint8_t J4[2];
    for (size_t p = 0; p < num_polarizations; ++p) {
      J4[p] = (uint32_t(clamp(J[p][0], -7, 7)) << 4) | uint32_t(clamp(J[p][1], -7, 7));
    }
    // Combine polarizations and add bias
    // TODO: Use make_uchar2?
    const uint16_t J4all = (uint32_t(J4[0]) | (uint32_t(J4[1]) << 8)) ^ 0x8888U;
    J_shared[beam][time % J_shared_time_modulo] = J4all;
  }
}
} // namespace reduce_to_J

using reduce_to_J::J_shared_time_modulo;

using reduce_to_J::J_shared_t;

////////////////////////////////////////////////////////////////////////////////

namespace transpose_J {

constexpr size_t num_beam_iterations = 3;
constexpr size_t num_beam_warps = 32;
constexpr size_t num_time_threads = 16;
constexpr size_t num_polarization_threads = 2;
constexpr size_t num_time_explicit = 4;
constexpr size_t num_complex_explicit = 2;

static_assert(num_beam_warps == num_warps);
static_assert(num_time_threads * num_polarization_threads == num_threads);
static_assert(num_time_explicit * num_complex_explicit == num_reals_int);
static_assert(num_time_iterations_outer * num_time_threads * num_time_explicit == num_times);
static_assert(num_beam_iterations * num_beam_warps == num_beams);

__device__ void transpose_J(ucomplex4 *restrict const J_array, const J_shared_t &restrict J_shared, const size_t frequency,
                            const size_t time_iteration_outer) {
  const size_t beam_warp = threadIdx.y;
  const size_t time_thread = threadIdx.x / num_polarization_threads;
  const size_t polarization_thread = threadIdx.x % num_polarization_threads;
  const size_t time0 = (time_iteration_outer * num_time_threads + time_thread) * num_time_explicit;
  const size_t polarization = polarization_thread;
  for (size_t beam_iteration = 0; beam_iteration < num_beam_iterations; ++beam_iteration) {
    const size_t beam = beam_iteration * num_beam_warps + beam_warp;
    // Load data
    // (We load twice as much as we need from shared memory)
    // (We could avoid bank conflicts here by interchanging shared memory reads on every other thread)
    uint32_t Jall0[2];
    Jall0[0] = *(const uint32_t *)&J_shared[beam][(time0 + 0 * num_time_explicit / 2) % J_shared_time_modulo];
    Jall0[1] = *(const uint32_t *)&J_shared[beam][(time0 + 1 * num_time_explicit / 2) % J_shared_time_modulo];
    // Extract polarization
    const uint32_t Jall1 = __byte_perm(Jall0[0], Jall0[1], polarization == 0 ? 0x6420 : 0x7531);
    // Write to global memory
    *(uint32_t *)&J_array[Jlinear(beam, frequency, polarization, time0, 0) / 2] = Jall1;
  }
}
} // namespace transpose_J

////////////////////////////////////////////////////////////////////////////////

__global__ void __launch_bounds__(num_threads *num_warps, 1)
    form_beams(ucomplex4 *restrict const J_array, const ucomplex4 *restrict const E_array, const ucomplex4 *restrict const A_array,
               const float *restrict const G_array) {

  // Each frequency is transformed independently. We use one thread block per frequency.

  const size_t frequency = blockIdx.x;

  // Load A into registers
  load_A::A_register_t A_register;
  load_A::load_A(A_register, A_array, frequency);

  for (size_t time_iteration_outer = 0; time_iteration_outer < num_time_iterations_outer; ++time_iteration_outer) {
    __shared__ E_shared_t E_shared;
    __shared__ Ju_shared_t Ju_shared;
    __shared__ J_shared_t J_shared;

    for (size_t time_iteration_inner = 0; time_iteration_inner < num_time_iterations_inner; ++time_iteration_inner) {
      shuffle_E::shuffle_E(E_shared, E_array, frequency, time_iteration_outer, time_iteration_inner);
      __syncthreads();

      for (size_t time_iteration_inner2 = 0; time_iteration_inner2 < num_time_iterations_inner2; ++time_iteration_inner2) {
        compute_Ju::compute_Ju(Ju_shared, A_register, E_shared, G_array, frequency, time_iteration_outer, time_iteration_inner,
                               time_iteration_inner2);
        __syncthreads();
        reduce_to_J::reduce_to_J(J_shared, Ju_shared, time_iteration_outer, time_iteration_inner, time_iteration_inner2);
      }
    }
    __syncthreads();
    transpose_J::transpose_J(J_array, J_shared, frequency, time_iteration_outer);
  }
}

////////////////////////////////////////////////////////////////////////////////

#define CHECK_RESULT(err) check_result(__FILE__, __LINE__, err)
void check_result(const char *file, int line, hipError_t err) {
  if (err != hipSuccess) {
    cerr << file << ":" << line << ": CUDA error " << err << ": " << hipGetErrorName(err) << ": " << hipGetErrorString(err)
         << "\n";
    exit(1);
  }
}

int main(int argc, char **argv) {
  cout << "beamforming.cuda4\n";

  vector<ucomplex4> Earray;
  vector<ucomplex4> Aarray;
  vector<float> Garray;
  vector<ucomplex4> Jarray;
  setup(Earray, Aarray, Garray, Jarray);

  cout << "Forming beams...\n";
  ucomplex4 *Earray2 = nullptr;
  hipMalloc(&Earray2, Earray.size() * sizeof(ucomplex4));
  hipMemcpy(Earray2, Earray.data(), Earray.size() * sizeof(ucomplex4), hipMemcpyHostToDevice);
  ucomplex4 *Aarray2 = nullptr;
  hipMalloc(&Aarray2, Aarray.size() * sizeof(ucomplex4));
  hipMemcpy(Aarray2, Aarray.data(), Aarray.size() * sizeof(ucomplex4), hipMemcpyHostToDevice);
  float *Garray2 = nullptr;
  hipMalloc(&Garray2, Garray.size() * sizeof(float));
  hipMemcpy(Garray2, Garray.data(), Garray.size() * sizeof(float), hipMemcpyHostToDevice);
  ucomplex4 *Jarray2 = nullptr;
  hipMalloc(&Jarray2, Jarray.size() * sizeof(ucomplex4));

  hipError_t err = hipGetLastError();
  CHECK_RESULT(err);

  const auto t0 = gettime();

  const dim3 numBlocks(num_frequencies);
  const dim3 threadsPerBlock(num_threads, num_warps);
  form_beams<<<numBlocks, threadsPerBlock>>>(Jarray2, Earray2, Aarray2, Garray2);
  err = hipGetLastError();
  CHECK_RESULT(err);
  err = hipDeviceSynchronize();
  CHECK_RESULT(err);

  const auto t1 = gettime();
  cout << "Elapsed time: " << (t1 - t0) << " seconds\n";

  err = hipGetLastError();
  CHECK_RESULT(err);

  hipFree(Earray2);
  Earray2 = nullptr;
  hipFree(Aarray2);
  Aarray2 = nullptr;
  hipFree(Garray2);
  Garray2 = nullptr;
  hipMemcpy(Jarray.data(), Jarray2, Jarray.size() * sizeof(ucomplex4), hipMemcpyDeviceToHost);
  hipFree(Jarray2);
  Jarray2 = nullptr;

  check(Jarray);

  cout << "Done.\n";
  return 0;
}
