#include "hip/hip_runtime.h"
// -*-c++-*-
// CUDA fragment layout

#include <mma.h>

#include <cassert>
#include <iostream>
#include <vector>

using namespace std;

using namespace nvcuda;
using namespace nvcuda::wmma;

// These sizes are dictated by CUDA
constexpr int m = 8;
constexpr int n = 8;
constexpr int k = 32;

// A[m][k]   must be row major
// B[k][n]   must be column major
// C[m][n]   row major
// C[m][n] += A[m][k] * B[k][n]

__global__ void matmul(int *restrict const Cptr) {
  fragment<wmma::accumulator, m, n, k, int> C;

  // Set fragment
  for (int t = 0; t < C.num_elements; t++)
    if (t % 2 == 0)
      C.x[t] = 2 * threadIdx.x + 0;
    else
      C.x[t] = 2 * threadIdx.x + 1;

  // Store to global memory
  store_matrix_sync(Cptr, C, n, mem_row_major);

  // // Store to local memory (not supported by wmma)
  // int Ctmp[m * n];
  // store_matrix_sync(Ctmp, C, n, mem_row_major);

  // Cptr[2 * threadIdx.x + 0] = Ctmp[2 * threadIdx.x + 0];
  // Cptr[2 * threadIdx.x + 1] = Ctmp[2 * threadIdx.x + 1];
}

__global__ void matmul2(int *restrict const Cptr) {
  fragment<wmma::accumulator, m, n, k, int> C;
  // Set fragment
  for (int t = 0; t < C.num_elements; t++)
    C.x[t] = 0;

  fragment<wmma::matrix_a, m, n, k, experimental::precision::s4, row_major> A;
  // Set fragment
  static_assert(A.num_elements == 8 * A.num_storage_elements, "");
  for (int t = 0; t < A.num_storage_elements; ++t)
    A.x[t] = (((8 * threadIdx.x + 7) & 0x0f) << 0x1c) |
             (((8 * threadIdx.x + 6) & 0x0f) << 0x18) |
             (((8 * threadIdx.x + 5) & 0x0f) << 0x14) |
             (((8 * threadIdx.x + 4) & 0x0f) << 0x10) |
             (((8 * threadIdx.x + 3) & 0x0f) << 0x0c) |
             (((8 * threadIdx.x + 2) & 0x0f) << 0x08) |
             (((8 * threadIdx.x + 1) & 0x0f) << 0x04) |
             (((8 * threadIdx.x + 0) & 0x0f) << 0x00);

  fragment<wmma::matrix_b, m, n, k, experimental::precision::s4, col_major> B;
  // Set fragment
  for (int t = 0; t < B.num_elements; t++)
    B.x[t] = 1;

  // Store to global memory
  mma_sync(C, A, B, C);
  store_matrix_sync(Cptr, C, n, mem_row_major);
}

#define CHECK_RESULT(err) check_result(__FILE__, __LINE__, err)
void check_result(const char *file, int line, hipError_t err) {
  if (err != hipSuccess) {
    cerr << file << ":" << line << ": CUDA error " << err << ": "
         << hipGetErrorName(err) << ": " << hipGetErrorString(err) << "\n";
    exit(1);
  }
}

int main(int argc, char **argv) {
  cout << "fragment.cuda\n";

  vector<int> Cvec(m * n);

  const auto Cget = [&](size_t r, size_t c) {
    assert(r < m && c < n);
    return Cvec.at(c + n * r);
  };

  int *Cptr = nullptr;
  hipMalloc(&Cptr, Cvec.size() * sizeof *Cvec.data());
  hipMemcpy(Cptr, Cvec.data(), Cvec.size() * sizeof *Cvec.data(),
             hipMemcpyHostToDevice);

  hipError_t err = hipGetLastError();
  CHECK_RESULT(err);
  const dim3 numBlocks(1);
  const dim3 threadsPerBlock(32);
  matmul<<<numBlocks, threadsPerBlock>>>(Cptr);
  err = hipGetLastError();
  CHECK_RESULT(err);
  err = hipDeviceSynchronize();
  CHECK_RESULT(err);
  err = hipGetLastError();
  CHECK_RESULT(err);

  hipMemcpy(Cvec.data(), Cptr, Cvec.size() * sizeof *Cvec.data(),
             hipMemcpyDeviceToHost);
  hipFree(Cptr);

  cout << "C:\n";
  for (size_t r = 0; r < m; ++r) {
    cout << "  ";
    for (size_t c = 0; c < n; ++c) {
      cout << Cget(r, c) << " ";
    }
    cout << "\n";
  }

  cout << "Done.\n";
  return 0;
}
